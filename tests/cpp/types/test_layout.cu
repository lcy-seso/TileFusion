#include "hip/hip_runtime.h"
// Copyright (c) Microsoft Corporation. All rights reserved.
// Licensed under the MIT License.

#include "common/test_utils.hpp"
#include "types/mod.hpp"

#include <iostream>

namespace tilefusion::testing {

using namespace tilefusion::cell;
namespace tl = tile_layout;

// TEST(TestLayout, test_layout) {
//     using Element = __half;

//     using Layout1 = tl::RowMajor<4, 7>;
//     EXPECT_EQ(tl::num_rows<Layout1>, 4);
//     EXPECT_EQ(tl::num_cols<Layout1>, 7);
//     EXPECT_EQ(tl::get_numel<Layout1>, 28);
//     EXPECT_EQ(tl::row_stride<Layout1>, 7);
//     EXPECT_EQ(tl::col_stride<Layout1>, 1);

//     tl::Layout type1 = tl::layout_type<Layout1>;
//     EXPECT_EQ(type1, tl::Layout::kRowMajor);
//     auto layout_name1 = layout_type_to_str(type1);
//     EXPECT_EQ(layout_name1, "RowMajor");

//     using Layout2 = tl::ColMajor<4, 7>;
//     EXPECT_EQ(tl::num_rows<Layout2>, 4);
//     EXPECT_EQ(tl::num_cols<Layout2>, 7);
//     EXPECT_EQ(tl::get_numel<Layout2>, 28);
//     EXPECT_EQ(tl::row_stride<Layout2>, 1);
//     EXPECT_EQ(tl::col_stride<Layout2>, 4);

//     tl::Layout type2 = tl::layout_type<Layout2>;
//     EXPECT_EQ(type2, tl::Layout::kColMajor);
//     auto layout_name2 = layout_type_to_str(type2);
//     EXPECT_EQ(layout_name2, "ColMajor");
// }

TEST(TestLayout, test_block_layout) {
    using Layout = tl::BlockRowMajor<tl::RowMajor<4, 9>, tl::RowMajor<2, 3>>;

    EXPECT_EQ(Layout::kTileRows, 2);
    EXPECT_EQ(Layout::kTileCols, 3);
    EXPECT_EQ(Layout::kRowStride, 18);
    EXPECT_EQ(Layout::kColStride, 6);
    EXPECT_EQ(Layout::kType, tl::Layout::kRowMajor);

    std::cout << "Strides: " << Layout::kRowStride << ", " << Layout::kColStride
              << std::endl;

    Layout layout;
    // int offset = layout(2, 6);
    // std::cout << "offset: " << offset << std::endl;

    // std::cout << "layout(2, 7): " << layout(2, 7) << std::endl;

    for (int i = 0; i < Layout::kRows; ++i) {
        for (int j = 0; j < Layout::kCols; ++j) {
            std::cout << layout(i, j) << ", ";
        }
        std::cout << std::endl;
    }
}

}  // namespace tilefusion::testing
