#include "hip/hip_runtime.h"
// Copyright (c) Microsoft Corporation. All rights reserved.
// Licensed under the MIT License.

#include "common/test_utils.hpp"
#include "types/mod.hpp"

#include <glog/logging.h>
#include <thrust/host_vector.h>

namespace tilefusion::testing {

using namespace cell;
namespace tl = tile_layout;

namespace {
template <typename Layout, typename ChunkShape, const tl::Layout kType>
struct GTileIteratorTester;

template <typename Layout_, typename ChunkShape>
struct GTileIteratorTester<Layout_, ChunkShape, tl::Layout::kRowMajor> {
    using Element = float;
    using Layout = Layout_;

    static constexpr int kRows = Layout::kRows;
    static constexpr int kCols = Layout::kCols;

    static constexpr int kStride0 = dim_size<0, ChunkShape>;
    static constexpr int kStride1 = dim_size<1, ChunkShape>;

    const int kTileRowStride = kStride0 * Layout::kRowStride;
    const int kTileColStride = kStride1;

    static_assert(kRows % kStride0 == 0, "kRows must be divisible by kStride0");
    static_assert(kCols % kStride1 == 0, "kCols must be divisible by kStride1");

    using Tile = GlobalTile<Element, tl::RowMajor<kRows, kCols>>;
    using Iterator = GTileIterator<Tile, ChunkShape>;

    void operator()() {
        int numel = kRows * kCols;
        thrust::host_vector<Element> data(numel);

        Layout layout;
        Element* ptr = data.data();
        int count = 0;
        for (int i = 0; i < kRows; ++i)
            for (int j = 0; j < kCols; ++j) ptr[count++] = layout(i, j);

#if defined(DEBUG_PRINT)
        Tile gtile(ptr);
        gtile.dump_value();
#endif

        EXPECT_EQ(Iterator::sc0, kRows / kStride0);
        EXPECT_EQ(Iterator::sc1, kCols / kStride1);

        Iterator iter(data.data());

        for (int i = 0; i < Iterator::sc0; ++i) {
            for (int j = 0; j < Iterator::sc1; ++j) {
                int start_n = i * kTileRowStride + j * kTileColStride;
                auto tile = iter(i, j);
                for (int m = 0; m < kStride0; ++m) {
                    for (int n = 0; n < kStride1; ++n) {
                        int v1 = int(tile(m, n));
                        int v2 = start_n + m * Layout::kRowStride + n;
                        EXPECT_EQ(v1, v2);
                    }
                }

#if defined(DEBUG_PRINT)
                printf("\nIteration-[%d, %d]:\n", i, j);
                iter(i, j).dump_value();
                printf("\n");
#endif
            }
        }
    }
};

template <typename Layout_, typename ChunkShape>
struct GTileIteratorTester<Layout_, ChunkShape, tl::Layout::kColMajor> {
    using Element = float;
    using Layout = Layout_;

    static constexpr int kRows = Layout::kRows;
    static constexpr int kCols = Layout::kCols;

    static constexpr int kStride0 = dim_size<0, ChunkShape>;
    static constexpr int kStride1 = dim_size<1, ChunkShape>;

    const int kTileRowStride = kStride0;
    const int kTileColStride = kStride1 * Layout::kColStride;

    static_assert(kRows % kStride0 == 0, "kRows must be divisible by kStride0");
    static_assert(kCols % kStride1 == 0, "kCols must be divisible by kStride1");

    using Tile = GlobalTile<Element, tl::RowMajor<kRows, kCols>>;
    using Iterator = GTileIterator<Tile, ChunkShape>;

    void operator()() {
        int numel = kRows * kCols;
        thrust::host_vector<Element> data(numel);

        Layout layout;
        Element* ptr = data.data();
        int count = 0;
        for (int i = 0; i < kRows; ++i)
            for (int j = 0; j < kCols; ++j) ptr[count++] = layout(i, j);

#if defined(DEBUG_PRINT)
        Tile gtile(ptr);
        gtile.dump_value();
#endif

        EXPECT_EQ(Iterator::sc0, kRows / kStride0);
        EXPECT_EQ(Iterator::sc1, kCols / kStride1);

        Iterator iter(data.data());

        for (int i = 0; i < Iterator::sc0; ++i) {
            for (int j = 0; j < Iterator::sc1; ++j) {
                int start_n = i * kTileRowStride + j * kTileColStride;

                auto tile = iter(i, j);
                for (int m = 0; m < kStride0; ++m) {
                    for (int n = 0; n < kStride1; ++n) {
                        int v1 = int(tile(m, n));
                        int v2 = start_n + m + n * Layout::kColStride;

                        EXPECT_EQ(v1, v2);
                    }
                }

#if defined(DEBUG_PRINT)
                printf("\nIteration-[%d, %d]:\n", i, j);
                iter(i, j).dump_value();
                printf("\n");
#endif
            }
        }
    }
};

__device__ void init_buf(void* buf, int numel) {
    for (int i = 0; i < numel; ++i) {
        reinterpret_cast<int*>(buf)[i] = i;
    }
}

template <typename Shared, typename SIterator>
__global__ void test_shared_tile_iterator() {
    using DType = typename Shared::DType;
    extern __shared__ __align__(sizeof(double)) unsigned char buf_[];
    DType* buf = reinterpret_cast<DType*>(buf_);

    init_buf(buf, Shared::kNumel);

    Shared s_tile(buf);

    s_tile.dump_value();
}

}  // namespace

// TEST(TestGTileIterator, test_row_major) {
//     using Tester = GTileIteratorTester<tl::RowMajor<4, 9>, TileShape<2, 3>,
//                                        tl::Layout::kRowMajor>;
//     Tester tester;
//     tester();
// }

// TEST(TestGTileIterator, col_major) {
//     using Tester = GTileIteratorTester<tl::ColMajor<4, 9>, TileShape<2, 3>,
//                                        tl::Layout::kColMajor>;
//     Tester tester;
//     tester();
// }

TEST(TestSharedTileIterator, row_major) {
    using InType = __half;
    static constexpr int kRows = 16;
    static constexpr int kCols = 64;

    static constexpr int kChunkRows = 8;
    static constexpr int kChunkCols = 64;

    using SharedLayout = tl::BlockRowMajor<
        tl::RowMajor<kRows, kCols>,
        SwizzledLayout<tl::RowMajor<8, 64>, Swizzle<3, 3, 3>>>;

    using Shared = SharedTile<InType, SharedLayout>;
    using SIterator = STileIterator<Shared, TileShape<kChunkRows, kChunkCols>>;

    LOG(INFO) << std::endl << Shared{} << std::endl;
    LOG(INFO) << std::endl << SIterator{} << std::endl;

    test_shared_tile_iterator<Shared, SIterator><<<1, 1>>>();
}
}  // namespace tilefusion::testing
