#include "hip/hip_runtime.h"
// Copyright (c) Microsoft Corporation. All rights reserved.
// Licensed under the MIT License.

#include "kernels/fused_two_gemms_device.cuh"
#include "util.hpp"

using namespace tilefusion::kernels;
namespace tl = tilefusion::tile_layout;

// kernel wrapper
template <typename InType, typename AccType, typename Config>
__attribute__((global)) void kernel_wrapper(const InType* A, const InType* B,
                                            const InType* C, InType* D) {
    ke_fused_two_gemms<InType, AccType, Config>(A, B, C, D);
}

template <typename WholeShape, typename CtaTileShape, typename WarpLayout,
          const int kBatch, const int kSharedAccess>
void run(float epsilon = 1e-3) {
    using InType = __half;
    using AccType = float;

    static constexpr int kM = dim_size<0, WholeShape>;
    static constexpr int kN = dim_size<1, WholeShape>;
    static constexpr int kK = dim_size<2, WholeShape>;
    static constexpr int kP = dim_size<3, WholeShape>;

    static constexpr int kTM = dim_size<0, CtaTileShape>;
    static constexpr int kTN = dim_size<1, CtaTileShape>;
    static constexpr int kTK = dim_size<2, CtaTileShape>;
    static constexpr int kTP = dim_size<3, CtaTileShape>;

    static_assert(kK == kTK, "The current implementation requires kTK == K.");
    static_assert(kP == kTP, "The current implementation requires kTP == P.");

    thrust::host_vector<InType> h_a(kM * kK * kBatch);

    for (int i = 0; i < h_a.size(); ++i) {
        h_a[i] = static_cast<InType>(rand_float());
    }

    thrust::host_vector<InType> h_b(kK * kN * kBatch);
    for (int i = 0; i < h_b.size(); ++i) {
        h_b[i] = static_cast<InType>(rand_float());
    }

    thrust::host_vector<InType> h_c(kN * kP * kBatch);
    for (int i = 0; i < h_c.size(); ++i) {
        h_c[i] = static_cast<InType>(rand_float());
    }

    thrust::host_vector<InType> h_d(kM * kP * kBatch);
    thrust::fill(h_d.begin(), h_d.end(), 0.);

    thrust::device_vector<InType> d_a = h_a;
    thrust::device_vector<InType> d_b = h_b;
    thrust::device_vector<InType> d_c = h_c;
    thrust::device_vector<InType> d_d = h_d;

    const InType* A = thrust::raw_pointer_cast(d_a.data());
    const InType* B = thrust::raw_pointer_cast(d_b.data());
    const InType* C = thrust::raw_pointer_cast(d_c.data());
    InType* D = thrust::raw_pointer_cast(d_d.data());

    using Config = FusedTwoGemmsTraits<InType, AccType, WarpLayout, kM, kN, kK,
                                       kP, kTM, kTN, kTK, kTP>;

    int block_x = CeilDiv<kM, kTM>;
    int block_y = CeilDiv<kP, kTP>;
    int block_z = kBatch;
    dim3 grid(block_x, block_y, block_z);

    static constexpr int kThreads = tl::get_numel<WarpLayout> * 32;
    dim3 block(kThreads, 1, 1);

    static constexpr int kShmInput = (kTM * kTK + kTK * kTN + kTN * kTP);
    static constexpr int kShmOutput = kTM * kTP;
    static constexpr int kSharedSize = kShmInput < kShmOutput
                                           ? kShmOutput * sizeof(InType)
                                           : kShmInput * sizeof(InType);

    auto kernel = &kernel_wrapper<InType, AccType, Config>;

    // FIXME(ying): make the hard-coded shared memory size dependent on the
    // underlying hardware through `hipGetDeviceProperties`
    if (kSharedSize > 48 * 1024) {
        hipFuncSetAttribute(reinterpret_cast<const void*>(
            kernel), hipFuncAttributeMaxDynamicSharedMemorySize, kSharedSize);
    }

    kernel<<<grid, block, kSharedSize, 0>>>(A, B, C, D);
    hipDeviceSynchronize();

    h_d = d_d;

    thrust::host_vector<InType> h_acc(kM * kN * kBatch);
    thrust::fill(h_acc.begin(), h_acc.end(), 0.);
    thrust::device_vector<InType> d_acc = h_acc;

    thrust::host_vector<InType> h_d2(kM * kP * kBatch);
    thrust::fill(h_d2.begin(), h_d2.end(), 0.);
    thrust::device_vector<InType> d_d2 = h_d2;

    cublas_two_gemms(kM, kN, kK, kP, kBatch, A, B, C,
                     thrust::raw_pointer_cast(d_d2.data()),
                     thrust::raw_pointer_cast(d_acc.data()));
    hipDeviceSynchronize();
    h_acc = d_acc;
    h_d2 = d_d2;

    InType* data = thrust::raw_pointer_cast(h_d.data());
    __half* ground_truth = thrust::raw_pointer_cast(h_d2.data());

#if 0
    int cut_off = 128;
    cut_off = cut_off > h_d.size() ? h_d.size() : cut_off;
    printf("ours:\n");
    for (int i = 0; i < cut_off; ++i) {
        printf("%.3f, ", __half2float(data[i]));
        if (i && (i + 1) % 16 == 0) printf("\n");
    }
    printf("\nground_truth:\n");
    for (int i = 0; i < cut_off; ++i) {
        printf("%.3f, ", __half2float(ground_truth[i]));
        if (i && (i + 1) % 16 == 0) printf("\n");
    }
#endif

    if (check_results(data, ground_truth, kM * kP, epsilon)) {
        std::cout << "[" << kM << ", " << kN << ", " << kK << ", " << kP
                  << "], batch = " << kBatch << ", passed." << std::endl;
    } else {
        std::cout << "[" << kM << ", " << kN << ", " << kK << ", " << kP
                  << "], batch = " << kBatch << ", failed." << std::endl;
    }
}

int main() {
    using WarpLayout = tl::RowMajor<2, 1>;
    static constexpr int kSharedAccess = 64;

    run<B2BGemmShape<256 /*M*/, 128 /*N*/, 64 /*K*/, 64 /*P*/>,
        B2BGemmShape<64 /*kTM*/, 64 /*kTN*/, 64 /*kTK*/, 64 /*kTP*/>,
        WarpLayout, 1, kSharedAccess>(5e-3);

    return 0;
}
