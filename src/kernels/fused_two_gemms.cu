#include "hip/hip_runtime.h"
// Copyright (c) Microsoft Corporation. All rights reserved.
// Licensed under the MIT License.

#include "cell/mod.hpp"
#include "jit/mod.hpp"
#include "kernels/common.hpp"
#include "kernels/ops.hpp"

namespace tilefusion::kernels {

using namespace tilefusion;
namespace tl = tile_layout;

std::string generate_fused_two_gemms_kernel_source(const std::string& in_type,
                                                   const std::string& acc_type,
                                                   int m, int n, int k, int p) {
    std::stringstream ss;
    ss << R"(
#include "kernels/fused_two_gemms_device.cuh"

extern "C" __global__ void fused_two_gemms_kernel_)"
       << in_type << "_" << acc_type << "_" << m << "_" << n << "_" << k << "_"
       << p << R"((
    const )"
       << in_type << R"(* A,
    const )"
       << in_type << R"(* B,
    const )"
       << in_type << R"(* C,
    )" << in_type
       << R"(* D,
    int m, int n, int k, int p) {
    using Config = tilefusion::kernels::FusedTwoGemmsTraits<)"
       << in_type << ", " << acc_type << R"(,
        tl::RowMajor<2, 1>, )"
       << m << ", " << n << ", " << k << ", " << p << R"(>;
})";
    return ss.str();
}

void fused_two_gemms(const torch::Tensor& A, const torch::Tensor& B,
                     const torch::Tensor& C, torch::Tensor& D) {
    CHECK_INPUT(A);
    CHECK_INPUT(B);
    CHECK_INPUT(C);
    CHECK_INPUT(D);

    const at::ScalarType dtype = A.scalar_type();
    TORCH_CHECK(dtype == at::ScalarType::Half && B.scalar_type() == dtype &&
                    C.scalar_type() == dtype && D.scalar_type() == dtype,
                "the inputs and output must be half-precision (fp16).");

    const int64_t m = A.size(0);
    const int64_t n = B.size(1);
    const int64_t k = B.size(1);
    const int64_t p = C.size(1);

    using WarpLayout = tl::RowMajor<2, 1>;
    using InType = __half;
    using AccType = float;

    std::string in_type = jit::get_type_string<InType>();
    std::string acc_type = jit::get_type_string<AccType>();

    std::string kernel_source =
        generate_fused_two_gemms_kernel_source(in_type, acc_type, m, n, k, p);

    std::string kernel_name = "fused_two_gemms_kernel_" + in_type + "_" +
                              acc_type + "_" + std::to_string(m) + "_" +
                              std::to_string(n) + "_" + std::to_string(k) +
                              "_" + std::to_string(p);

    auto& jit = jit::JitCompiler::instance();

    // Get the project root directory from the current file's location
    std::string current_file = __FILE__;
    std::string project_root =
        current_file.substr(0, current_file.find("/src/"));

    std::vector<std::string> include_paths = {
        project_root + "/include", project_root + "/3rd-party/cutlass/include"};

    std::vector<std::string> compile_args = {"-O3",
                                             "-std=c++20",
                                             "--expt-relaxed-constexpr",
                                             "--expt-extended-lambda",
                                             "-DNDEBUG",
                                             "-Xcompiler",
                                             "-fPIC",
                                             "-Xcompiler",
                                             "-Wall",
                                             "-Xcompiler",
                                             "-Wextra"};
    hipFunction_t kernel = jit.get_or_compile_kernel(kernel_name, kernel_source,
                                                  include_paths, compile_args);

    if (!kernel) {
        throw std::runtime_error("Failed to compile or retrieve kernel");
    }

    // FIXME(ying): this should be tuned properly for the best performance
    int block_size = 128;
    int grid_size = (m * p + block_size - 1) / block_size;

    const InType* A_ptr =
        reinterpret_cast<const InType*>(A.data_ptr<at::Half>());
    const InType* B_ptr =
        reinterpret_cast<const InType*>(B.data_ptr<at::Half>());
    const InType* C_ptr =
        reinterpret_cast<const InType*>(C.data_ptr<at::Half>());
    InType* D_ptr = reinterpret_cast<InType*>(D.data_ptr<at::Half>());

    void* args[] = {(void*)&A_ptr, (void*)&B_ptr, (void*)&C_ptr, (void*)&D_ptr,
                    (void*)&m,     (void*)&n,     (void*)&k,     (void*)&p};

    CUDA_DRIVER_CHECK(hipModuleLaunchKernel(kernel, grid_size, 1, 1, block_size, 1, 1,
                                     0, nullptr, args, nullptr));

    LOG(INFO) << "Fused two gemms kernel launched successfully";
}
}  // namespace tilefusion::kernels
