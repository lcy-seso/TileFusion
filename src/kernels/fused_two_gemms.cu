#include "hip/hip_runtime.h"
// Copyright (c) Microsoft Corporation. All rights reserved.
// Licensed under the MIT License.

#include "cell/mod.hpp"
#include "jit/mod.hpp"
#include "kernels/common.hpp"
#include "kernels/ops.hpp"

namespace tilefusion::kernels {

using namespace tilefusion;
namespace tl = tile_layout;

std::string generate_fused_two_gemms_kernel_source(const std::string& in_type,
                                                   const std::string& acc_type,
                                                   int m, int n, int k, int p) {
    std::stringstream ss;
    ss << R"(
#include "kernels/fused_two_gemms_device.cuh"

extern "C" __global__ void fused_two_gemms_kernel_)"
       << in_type << "_" << acc_type << "_" << m << "_" << n << "_" << k << "_"
       << p << R"((
    const )"
       << in_type << R"(* A,
    const )"
       << in_type << R"(* B,
    const )"
       << in_type << R"(* C,
    )" << in_type
       << R"(* D,
    int m, int n, int k, int p) {
    using Config = FusedTwoGemmsTraits<)"
       << in_type << ", " << acc_type << R"(,
        tl::RowMajor<2, 1>, )"
       << m << ", " << n << ", " << k << ", " << p << R"(>;
})";
    return ss.str();
}

void fused_two_gemms(const torch::Tensor& A, const torch::Tensor& B,
                     const torch::Tensor& C, torch::Tensor& D) {
    CHECK_INPUT(A);
    CHECK_INPUT(B);
    CHECK_INPUT(C);
    CHECK_INPUT(D);

    const at::ScalarType dtype = A.scalar_type();
    TORCH_CHECK(dtype == at::ScalarType::Half && B.scalar_type() == dtype &&
                    C.scalar_type() == dtype && D.scalar_type() == dtype,
                "the inputs and output must be half-precision (fp16).");

    const int64_t m = A.size(0);
    const int64_t n = B.size(1);
    const int64_t k = B.size(1);
    const int64_t p = C.size(1);

    using WarpLayout = tl::RowMajor<2, 1>;
    using InType = __half;
    using AccType = float;

    std::string in_type = jit::get_type_string<InType>();
    std::string acc_type = jit::get_type_string<AccType>();

    // Generate kernel source
    std::string kernel_source =
        generate_fused_two_gemms_kernel_source(in_type, acc_type, m, n, k, p);

    // Create unique kernel name
    std::string kernel_name = "fused_two_gemms_kernel_" + in_type + "_" +
                              acc_type + "_" + std::to_string(m) + "_" +
                              std::to_string(n) + "_" + std::to_string(k) +
                              "_" + std::to_string(p);

    auto& jit = jit::JitCompiler::instance();
    hipFunction_t kernel = jit.get_or_compile_kernel(kernel_name, kernel_source);

    if (!kernel) {
        throw std::runtime_error("Failed to compile or retrieve kernel");
    }

    int block_size = 128;  // Adjust based on your needs
    int grid_size = (m * p + block_size - 1) / block_size;

    // Properly cast the tensor data pointers to half precision
    const InType* A_ptr =
        reinterpret_cast<const InType*>(A.data_ptr<at::Half>());
    const InType* B_ptr =
        reinterpret_cast<const InType*>(B.data_ptr<at::Half>());
    const InType* C_ptr =
        reinterpret_cast<const InType*>(C.data_ptr<at::Half>());
    InType* D_ptr = reinterpret_cast<InType*>(D.data_ptr<at::Half>());

    void* args[] = {(void*)&A_ptr, (void*)&B_ptr, (void*)&C_ptr, (void*)&D_ptr,
                    (void*)&m,     (void*)&n,     (void*)&k,     (void*)&p};

    CUDA_DRIVER_CHECK(hipModuleLaunchKernel(kernel, grid_size, 1, 1, block_size, 1, 1,
                                     0, nullptr, args, nullptr));

    LOG(INFO) << "Fused two gemms kernel launched successfully";
}
}  // namespace tilefusion::kernels
