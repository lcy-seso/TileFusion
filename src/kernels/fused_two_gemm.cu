#include "hip/hip_runtime.h"
// Copyright (c) Microsoft Corporation. All rights reserved.
// Licensed under the MIT License.

#include "cell/mod.hpp"
#include "kernels/fused_two_gemms.hpp"
#include "types/mod.hpp"

using namespace tilefusion;
using namespace cell;
using namespace copy;
using namespace compute;
namespace tl = tile_layout;

namespace tilefusion::kernels {

namespace {

template <typename InType, typename AccType,                       //
          typename WarpLayout, const int kSharedAccess,            //
          const int kM, const int kN, const int kK, const int kP,  //
          const int kTM, const int kTN, const int kTK, const int kTP>
struct KeTraits {
    using BaseShape = traits::BaseTileShape<InType>;

    static constexpr int kWarpPerRow = tl::num_rows<WarpLayout>;
    static constexpr int kWarpPerCol = tl::num_cols<WarpLayout>;
    static_assert(kWarpPerCol == 1, "WarpPerCol must be 1");

    static constexpr int kThreads = tl::get_numel<WarpLayout> * 32;

    // operand A
    using GlobalA = GlobalTile<InType, tl::RowMajor<kTM, kK>>;
    // chunk the K dimension to fit into shared memory
    using GIteratorA = GTileIterator<GlobalA, TileShape<kTM, kTK>>;

    static const bool kUseSwizzling = true;

    using SharedA = SharedTile<InType, tl::RowMajor<kTM, kTK>, kUseSwizzling,
                               kSharedAccess>;

    static constexpr int kAMs = kTM / kWarpPerRow / BaseShape::kRows;
    static constexpr int kAKs = kTK / BaseShape::kCols;
    using RegA = RegTile<BaseTileRowMajor<InType>, tl::RowMajor<kAMs, kAKs>>;

    using SharedALoader = GlobalToSharedLoader<SharedA, WarpLayout>;
    using RegALoader =
        SharedToRegLoader<RegA, WarpLayout, WarpReuse::kRowReuseCont>;

    // operand B
    using GlobalB = GlobalTile<InType, tl::ColMajor<kK, kN>>;
    using GIteratorB = GTileIterator<GlobalB, TileShape<kTK, kTN>>;
    using SharedB = SharedTile<InType, tl::ColMajor<kTK, kTN>, kUseSwizzling,
                               kSharedAccess>;

    static constexpr int kBKs = kTK / BaseShape::kRows;
    static constexpr int kBNs = kTN / kWarpPerCol / BaseShape::kCols;
    using RegB = RegTile<BaseTileColMajor<InType>, tl::ColMajor<kBKs, kBNs>>;

    using SharedBLoader = GlobalToSharedLoader<SharedB, WarpLayout>;
    using RegBLoader =
        SharedToRegLoader<RegB, WarpLayout, WarpReuse::kColReuseCont>;

    // operand C
    using GlobalC = GlobalTile<InType, tl::ColMajor<kN, kTP>>;
    // chunk the N dimension to fit into shared memory
    using GIteratorC = GTileIterator<GlobalC, TileShape<kTN, kTP>>;
    using SharedC = SharedTile<InType, tl::ColMajor<kTN, kTP>, kUseSwizzling,
                               kSharedAccess>;

    static constexpr int kCNs = kTN / BaseShape::kRows;
    static constexpr int kCPs = kTP / kWarpPerCol / BaseShape::kCols;
    using RegC = RegTile<BaseTileColMajor<InType>, tl::ColMajor<kCNs, kCPs>>;

    using SharedCLoader = GlobalToSharedLoader<SharedC, WarpLayout>;
    using RegCLoader =
        SharedToRegLoader<RegC, WarpLayout, WarpReuse::kColReuseCont>;

    // output D
    using GlobalD = GlobalTile<InType, tl::RowMajor<kTM, kTP>>;
    using SharedD = SharedTile<InType, tl::RowMajor<kTM, kTP>, kUseSwizzling,
                               kSharedAccess>;

    static constexpr int kDMs = kTM / kWarpPerRow / BaseShape::kRows;
    static constexpr int kDPs = kTP / kWarpPerCol / BaseShape::kCols;
    using RegD = RegTile<BaseTileRowMajor<AccType>, tl::RowMajor<kDMs, kDPs>>;
    using RegDHalf =
        RegTile<BaseTileRowMajor<InType>, tl::RowMajor<kDMs, kDPs>>;

    static constexpr int kAccMs = kTM / kWarpPerRow / BaseShape::kRows;
    static constexpr int kAccNs = kTN / kWarpPerCol / BaseShape::kCols;

    // Reg Acc
    using RegAcc =
        RegTile<BaseTileRowMajor<AccType>, tl::RowMajor<kAccMs, kAccNs>>;
    using RegAccCast =
        RegTile<BaseTileRowMajor<InType>, tl::RowMajor<kAccMs, kAccNs>>;

    // Convert the accumulator to half
    using ConvertHalf = compute::RegTileConvert<RegAcc, RegAccCast>;
    using ConvertD = compute::RegTileConvert<RegD, RegDHalf>;

    using StoreRegD = RegToSharedStorer<RegDHalf, WarpLayout>;
    using StoreSharedD = SharedToGlobalStorer<SharedD, WarpLayout>;
};
}  // namespace

template <typename InType, typename AccType,                     //
          typename GIteratorA, typename SharedA, typename RegA,  //
          typename SharedALoader, typename RegALoader,           //
          typename GIteratorB, typename SharedB, typename RegB,  //
          typename SharedBLoader, typename RegBLoader,           //
          typename GIteratorC, typename SharedC, typename RegC,  //
          typename SharedCLoader, typename RegCLoader,           //
          typename RegAcc, typename RegAccCast, typename GlobalD,
          typename SharedD, typename RegD, typename RegDHalf,
          typename StoreRegD, typename StoreSharedD, typename ConvertAcc,
          typename ConvertD>
__global__ void ke_fused_two_gemms(const InType* dA, const InType* dB,
                                   const InType* dC, InType* dD, int kM, int kN,
                                   int kK, int kP, int kTM, int kTN, int kTK,
                                   int kTP) {
    // Advance to the global data tile to the current CTA.
    const InType* A = dA + blockIdx.z * (kM * kK) + blockIdx.x * (kTM * kK);
    const InType* B = dB + blockIdx.z * (kK * kN);
    const InType* gC_ptr =
        dC + blockIdx.z * (kN * kP) + blockIdx.y * (kTP * kN);

    InType* gD_ptr = dD + blockIdx.z * (kM * kP) + blockIdx.x * (kTM * kP) +
                     (blockIdx.y * kTP);

    extern __shared__ __align__(sizeof(double)) unsigned char shared_buf[];
    auto* shm = reinterpret_cast<InType*>(shared_buf);

    InType* sA_ptr = shm;
    InType* sB_ptr = shm + SharedA::kNumel;
    InType* sC_ptr = shm + SharedA::kNumel + SharedB::kNumel;
    InType* sD_ptr = shm;

    GIteratorA gAs(A);
    SharedA sA(sA_ptr);
    RegA rA;

    SharedALoader load_sa;
    RegALoader load_ra;

    GIteratorB gBs(B);
    SharedB sB(sB_ptr);
    RegB rB;

    SharedBLoader load_sb;
    RegBLoader load_rb;

    GIteratorC gCs(gC_ptr);
    SharedC sC(sC_ptr);

    SharedCLoader load_sc;
    RegCLoader load_rc;
    RegC rC;

    GlobalD gD(gD_ptr);
    SharedD sD(sD_ptr);
    RegD rD;
    RegDHalf rD_half;
    StoreRegD store_rD;
    StoreSharedD store_sD;

    RegAcc acc;
    RegAccCast acc_half;

    ConvertAcc cast_acc;  // Convert acc to half precision
    ConvertD convert_d;   // Convert D to half precision

    for (int n = 0; n < GIteratorC::sc0; ++n) {
        load_sc(gCs(n), sC);

        for (int k = 0; k < GIteratorA::sc1; ++k) {
            load_sa(gAs(k), sA);
            load_sb(gBs(k, n), sB);
            __copy_async();
            __syncthreads();

            load_ra(sA, rA);
            load_rb(sB, rB);
            __syncthreads();
            gemm(rA, rB, acc);
        }
        load_rc(sC, rC);
        __syncthreads();

        cast_acc(acc, acc_half);

        gemm(acc_half, rC, rD);
        acc.clear();
    }
    __syncthreads();
    convert_d(rD, rD_half);

    store_rD(rD_half, sD);
    __syncthreads();
    store_sD(sD, gD);
}

void fused_two_gemms(const torch::Tensor& A, const torch::Tensor& B,
                     const torch::Tensor& C, torch::Tensor& D) {
    // const int64_t m = A.size(0);
    // const int64_t n = B.size(1);
    // const int64_t k = B.size(1);
    // const int64_t p = C.size(1);

    using WarpLayout = tl::RowMajor<2, 1>;

    using InType = __half;
    using AccType = float;

    static constexpr int kSharedAccess = 64;

    // FIXME
    static constexpr int kM = 256;
    static constexpr int kN = 128;
    static constexpr int kK = 64;
    static constexpr int kP = 64;

    static constexpr int kTM = 64;
    static constexpr int kTN = 64;
    static constexpr int kTK = 64;
    static constexpr int kTP = 64;

    using Config = KeTraits<InType, AccType, WarpLayout, kSharedAccess,  //
                            kM, kN, kK, kP,      /* problem shape */
                            kTM, kTN, kTK, kTP>; /* cta tile shape */

    using RegA = typename Config::RegA;
    using RegB = typename Config::RegB;
    using RegC = typename Config::RegC;
    using RegD = typename Config::RegD;
    using RegDHalf = typename Config::RegDHalf;
    using RegAcc = typename Config::RegAcc;
    using RegAccCast = typename Config::RegAccCast;

    using GIteratorA = typename Config::GIteratorA;
    using SharedA = typename Config::SharedA;
    using SharedALoader = typename Config::SharedALoader;
    using RegALoader = typename Config::RegALoader;

    using GIteratorB = typename Config::GIteratorB;
    using SharedB = typename Config::SharedB;
    using SharedBLoader = typename Config::SharedBLoader;
    using RegBLoader = typename Config::RegBLoader;

    using GIteratorC = typename Config::GIteratorC;
    using SharedC = typename Config::SharedC;
    using SharedCLoader = typename Config::SharedCLoader;
    using RegCLoader = typename Config::RegCLoader;

    using SharedD = typename Config::SharedD;
    using StoreRegD = typename Config::StoreRegD;
    using StoreSharedD = typename Config::StoreSharedD;

    using ConvertAcc = typename Config::ConvertHalf;
    using ConvertD = typename Config::ConvertD;

    int block_x = CeilDiv<kM, kTM>;
    int block_y = CeilDiv<kP, kTP>;
    int block_z = 1;

    dim3 grid(block_x, block_y, block_z);
    dim3 block(Config::kThreads, 1, 1);

    int shm_input = (kTM * kTK + kTK * kTN + kTN * kTP);
    int shm_output = kTM * kTP;
    int shm_size = shm_input < shm_output ? shm_output * sizeof(InType)
                                          : shm_input * sizeof(InType);

    auto kernel =
        &ke_fused_two_gemms<InType, AccType,            //
                            GIteratorA, SharedA, RegA,  //
                            SharedALoader, RegALoader,  //
                            GIteratorB, SharedB, RegB,  //
                            SharedBLoader, RegBLoader,  //
                            GIteratorC, SharedC, RegC,  //
                            SharedCLoader, RegCLoader,  //
                            RegAcc, RegAccCast, typename Config::GlobalD,
                            SharedD, RegD, RegDHalf, StoreRegD, StoreSharedD,
                            ConvertAcc, ConvertD>;

    if (shm_size > 48 * 1024) {
        hipFuncSetAttribute(reinterpret_cast<const void*>(
            kernel), hipFuncAttributeMaxDynamicSharedMemorySize, shm_size);
    }

    kernel<<<grid, block, shm_size, 0>>>(
        A.data_ptr<InType>(), B.data_ptr<InType>(), C.data_ptr<InType>(),
        D.data_ptr<InType>(), kM, kN, kK, kP, kTM, kTN, kTK, kTP);
    hipDeviceSynchronize();
}

}  // namespace tilefusion::kernels
